#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define M 8192
#define N 2560
// #define M 128
// #define N 32
#define BLOCK_X 40
#define BLOCK_Y 32
#define TILE_SIZE 16

#define WARMUPS 100
#define ITERS 200

#define CHECK(call)                                          \
  {                                                          \
    const hipError_t error = call;                          \
    if (error != hipSuccess) {                              \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
      fprintf(stderr, "code: %d, reason: %s\n", error,       \
              hipGetErrorString(error));                    \
      exit(1);                                               \
    }                                                        \
  }

// elapsed time in millisecond
#define CpuElapse(base, start) \
  base += ((double)(clock() - start)) * 1000 / CLOCKS_PER_SEC;

#define GpuElapse(start, stop, elapsed, total) \
  hipEventRecord(stop, 0);                    \
  hipEventSynchronize(stop);                  \
  hipEventElapsedTime(&elapsed, start, stop); \
  total += elapsed;

// check result
// | (real - expected) / expected |
void check_result(float* host_ref, float* gpu_ref) {
  double epsilon = 1.0E-5;
  bool match = 1;
  for (int i = 0; i < N; i++) {
  gpu_ref[i] /= 300;

    if (abs((host_ref[i] - gpu_ref[i]) / host_ref[i]) > epsilon) {
      match = 0;
      printf("Arrays do not match!\n");
      printf("host %5.8f gpu %5.8f at index %d, error %5.8f\n", host_ref[i],
             gpu_ref[i], i, abs((host_ref[i] - gpu_ref[i]) / host_ref[i]));
      break;
    }
  }
  if (match) printf("Results match!\n");
}

// column reduction on host
void column_reduce_host(float* matrix, float* result) {
  for (int col = 0; col < N; col++) {
    float accum = 0.0;
    for (int row = 0; row < M; row++) {
      accum += matrix[row * N + col];
    }
    result[col] = accum;
  }
}

__device__ void warp_reduce(volatile float* sdata, int tid) {
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}

// colume reduction for a mtrix, 1D grid and 2D block
__global__ void column_reduce_trans(float* data_in, float* data_out) {
  __shared__ float sdata[BLOCK_X][BLOCK_Y];

  int block_x = blockIdx.x % ((N + BLOCK_X - 1) / BLOCK_X);
  int block_y = blockIdx.x / ((N + BLOCK_X - 1) / BLOCK_X);

  int col_g = block_x * blockDim.x + threadIdx.x;
  // int col = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid = threadIdx.x + threadIdx.y * blockDim.x;

  // local reduction
  float accum = 0.0f;
  for (int i = 0; i < TILE_SIZE; i++) {
    int row_g = i + threadIdx.y * TILE_SIZE + block_y * TILE_SIZE * BLOCK_Y;
    accum += data_in[row_g * N + col_g];
    // accum += data_in[(i + TILE_SIZE * threadIdx.y) * N + col];
  }
  sdata[threadIdx.x][threadIdx.y] = accum;
  __syncthreads();

  // printf("1 thread.x = %d, thread.y = %d, sdata[x][y] = %f\n", \
        threadIdx.x, threadIdx.y, sdata[threadIdx.x][threadIdx.y]);
  
  for (int stride = BLOCK_Y / 2; stride > 0; stride >>= 1) {
    if (threadIdx.y < stride) {
      sdata[threadIdx.x][threadIdx.y] +=
          sdata[threadIdx.x][threadIdx.y + stride];
    }
    __syncthreads();
  }

  // printf("2 blockIDx = %d, block_x= %d, block_y = %d,thread.x = %d, thread.y = %d, sdata[x][y] = %f\n", \
  //       blockIdx.x, block_x, block_y, threadIdx.x, threadIdx.y, sdata[threadIdx.x][threadIdx.y]);


  if (threadIdx.y == 0) {
    // data_out[col] = sdata[threadIdx.x][0];
    atomicAdd(&data_out[col_g], sdata[threadIdx.x][0]);
  }
}

int main() {
  // allocate memory for matrix and result on host
  float* matrix = (float*)malloc(M * N * sizeof(float));
  float* result = (float*)malloc(N * sizeof(float));
  float* h_result = (float*)malloc(N * sizeof(float));

  // initialize matrix
  for (int i = 0; i < M * N; i++) {
    // matrix[i] = (float)rand() / RAND_MAX;
    matrix[i] = 1.0f;
  }

  // allocate memory for matrix and result on device
  float *d_matrix, *d_result;
  CHECK(hipMalloc((void**)&d_matrix, M * N * sizeof(float)));
  CHECK(hipMalloc((void**)&d_result, N * sizeof(float)));
  CHECK(hipMemcpy(d_matrix, matrix, M * N * sizeof(float),
                   hipMemcpyHostToDevice));

  dim3 block(BLOCK_X, BLOCK_Y);
  //   dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  // dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) /
  // BLOCK_SIZE / TILE_SIZE);
  int grid_y = M / (TILE_SIZE * BLOCK_Y);
  dim3 grid((N + BLOCK_X - 1) / BLOCK_X * grid_y, 1);

  for (int i = 0; i < WARMUPS; ++i)
    column_reduce_trans<<<grid, block>>>(d_matrix, d_result);

  float total = 0.;
  float elapsed = 0.;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < ITERS; ++i)
    column_reduce_trans<<<grid, block>>>(d_matrix, d_result);

  GpuElapse(start, stop, elapsed, total);
  printf("column_reduce Time elapsed %f us\n", elapsed / ITERS * 1000);

  CHECK(
      hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipDeviceReset());

  // column reduction on host
  column_reduce_host(matrix, h_result);

  // check result
  check_result(h_result, result);

  free(matrix);
  free(result);
  free(h_result);
  hipFree(d_matrix);
  hipFree(d_result);

  return 0;
}
