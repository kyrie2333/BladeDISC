#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define M 8192
#define N 2560
// #define THREAD_ELEMENT_NUM 16
#define TILE_SIZE 256
#define BLOCK_SIZE 32

#define WARMUPS 100
#define ITERS 200

#define CHECK(call)                                          \
  {                                                          \
    const hipError_t error = call;                          \
    if (error != hipSuccess) {                              \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
      fprintf(stderr, "code: %d, reason: %s\n", error,       \
              hipGetErrorString(error));                    \
      exit(1);                                               \
    }                                                        \
  }

// elapsed time in millisecond
#define CpuElapse(base, start) \
  base += ((double)(clock() - start)) * 1000 / CLOCKS_PER_SEC;

#define GpuElapse(start, stop, elapsed, total) \
  hipEventRecord(stop, 0);                    \
  hipEventSynchronize(stop);                  \
  hipEventElapsedTime(&elapsed, start, stop); \
  total += elapsed;

// check result
void check_result(float* host_ref, float* gpu_ref) {
  double epsilon = 1.0E-5;
  bool match = 1;
  // | (real - expected) / expected |
  for (int i = 0; i < N; i++) {
    if (abs((host_ref[i] - gpu_ref[i]) / host_ref[i]) > epsilon) {
      match = 0;
      printf("Arrays do not match!\n");
      printf("host %5.8f gpu %5.8f at index %d, error %5.8f\n", host_ref[i], gpu_ref[i], i, abs((host_ref[i] - gpu_ref[i]) / host_ref[i]));
      break;
    }
  }
  if (match) printf("Results match.\n");
}

// column reduction on host
void column_reduce_host(float* matrix, float* result) {
  for (int col = 0; col < N; col++) {
    float accum = 0.0;
    for (int row = 0; row < M; row++) {
      accum += matrix[row * N + col];
    }
    result[col] = accum;
  }
}

__device__ void warp_reduce(volatile float* sdata, int tid) {
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}


// colume reduction for a mtrix, 1D grid and 2D block
__global__ void column_reduce_trans(float* data_in, float* data_out) {
  // shared memory for each block
  // __shared__ float sdata[BLOCK_SIZE*BLOCK_SIZE];
  __shared__ float sdata[BLOCK_SIZE][BLOCK_SIZE];

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid = threadIdx.x + threadIdx.y * blockDim.x;

  // local reduction
  float accum = 0.0f;
  for (int i = 0; i < TILE_SIZE; i++) {
    accum += data_in[(i + TILE_SIZE * threadIdx.y) * N + col];
  }
  // sdata[threadIdx.x * BLOCK_SIZE + threadIdx.y] = accum;
  sdata[threadIdx.x][threadIdx.y] = accum;
  __syncthreads();


  // printf("1 thread.x = %d, thread.y = %d, sdata[y][x] = %f\n", \
        threadIdx.x, threadIdx.y, sdata[threadIdx.y][threadIdx.x]);  
  warp_reduce(sdata[threadIdx.y], threadIdx.x);
  __syncthreads();
  // printf("2 thread.x = %d, thread.y = %d, sdata[y][x] = %f\n", \
        threadIdx.x, threadIdx.y, sdata[threadIdx.y][threadIdx.x]);

  // if (threadIdx.x % 32 == 0 ) {
  //     atomicAdd(&sdata[threadIdx.y][0], sdata[threadIdx.y][threadIdx.x]);
  // }

  // block reduction
  if (threadIdx.y % BLOCK_SIZE == 0) {
    data_out[blockIdx.x * BLOCK_SIZE + threadIdx.x] = sdata[threadIdx.x][0];
  }
}

int main() {
  // allocate memory for matrix and result on host
  float* matrix = (float*)malloc(M * N * sizeof(float));
  float* result = (float*)malloc(N * sizeof(float));
  float* h_result = (float*)malloc(N * sizeof(float));

  // initialize matrix
  for (int i = 0; i < M * N; i++) {
    matrix[i] = (float)rand() / RAND_MAX;
    // matrix[i] = 1.0f;
  }

  // allocate memory for matrix and result on device
  float *d_matrix, *d_result;
  CHECK(hipMalloc((void**)&d_matrix, M * N * sizeof(float)));
  CHECK(hipMalloc((void**)&d_result, N * sizeof(float)));
  CHECK(hipMemcpy(d_matrix, matrix, M * N * sizeof(float),
                   hipMemcpyHostToDevice));

  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  // dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) /
  // BLOCK_SIZE / TILE_SIZE);
  dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

  for (int i = 0; i < WARMUPS; ++i)
    column_reduce_trans<<<grid, block>>>(d_matrix, d_result);

  float total = 0.;
  float elapsed = 0.;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < ITERS; ++i)
    column_reduce_trans<<<grid, block>>>(d_matrix, d_result);

  GpuElapse(start, stop, elapsed, total);
  printf("column_reduce Time elapsed %f us\n", elapsed / ITERS * 1000);

  CHECK(
      hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipDeviceReset());

  // column reduction on host
  column_reduce_host(matrix, h_result);

  // check result
  check_result(h_result, result);

  free(matrix);
  free(result);
  free(h_result);
  hipFree(d_matrix);
  hipFree(d_result);

  return 0;
}
