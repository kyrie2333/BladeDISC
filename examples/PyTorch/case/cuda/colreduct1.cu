#include <hip/hip_runtime.h>
// #include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define M 8192
#define N 2560
#define THREAD_ELEMENT_NUM 16
#define BLOCK_SIZE 16

#define WARMUPS 100
#define ITERS 200

#define CHECK(call)                                          \
  {                                                          \
    const hipError_t error = call;                          \
    if (error != hipSuccess) {                              \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
      fprintf(stderr, "code: %d, reason: %s\n", error,       \
              hipGetErrorString(error));                    \
      exit(1);                                               \
    }                                                        \
  }

#define CpuElapse(base, start) \
  base += ((double)(clock() - start)) * 1000 / CLOCKS_PER_SEC;

#define GpuElapse(start, stop, elapsed, total) \
  hipEventRecord(stop, 0);                    \
  hipEventSynchronize(stop);                  \
  hipEventElapsedTime(&elapsed, start, stop); \
  total += elapsed;

// check result
void check_result(float* host_ref, float* gpu_ref) {
  double epsilon = 1.0E-5;
  bool match = 1;
  for (int i = 0; i < N; i++) {
    if (abs(host_ref[i] - gpu_ref[i]) > epsilon) {
      match = 0;
      printf("Arrays do not match!\n");
      printf("host %5.2f gpu %5.2f at index %d\n", host_ref[i], gpu_ref[i], i);
      break;
    }
  }
  if (match) printf("Results match.\n");
}

// column reduction on host
void column_reduce_host(float* matrix, float* result) {
  for (int col = 0; col < N; col++) {
    float accum = 0.0;
    for (int row = 0; row < M; row++) {
      accum += matrix[row * N + col];
    }
    result[col] = accum;
  }
}

// colume reduction for a mtrix, 2D grid and 1D block
// each thread will reduce a column in a block
// each accum of a block is reduce by atomicAdd
__global__ void column_reduce(float* data_in, float* data_out) {
  // get the column index
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float accum = 0.0;
  if (col < N) {
    for (int row = blockIdx.y * THREAD_ELEMENT_NUM;
         row < (blockIdx.y + 1) * THREAD_ELEMENT_NUM; row++) {
      accum += data_in[row * N + col];
    }
  }
  __syncthreads();

  atomicAdd(&data_out[col], accum);
}

int main() {
  // allocate memory for matrix and result on host
  float* matrix = (float*)malloc(M * N * sizeof(float));
  float* result = (float*)malloc(N * sizeof(float));    // gpu result
  float* h_result = (float*)malloc(N * sizeof(float));  // validation

  // initialize matrix
  for (int i = 0; i < M * N; i++) {
    matrix[i] = (float)rand() / RAND_MAX;
    // matrix[i] = 1.0;
  }

  // allocate memory for matrix and result on device
  float *d_matrix, *d_result;
  CHECK(hipMalloc((void**)&d_matrix, M * N * sizeof(float)));
  CHECK(hipMalloc((void**)&d_result, N * sizeof(float)));

  CHECK(hipMemcpy(d_matrix, matrix, M * N * sizeof(float),
                   hipMemcpyHostToDevice));

  dim3 block(BLOCK_SIZE, 1);
  dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, M / THREAD_ELEMENT_NUM);

  for (int i = 0; i < WARMUPS; ++i)
    column_reduce<<<grid, block>>>(d_matrix, d_result);

  float total = 0.;
  float elapsed = 0.;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < ITERS; ++i)
    column_reduce<<<grid, block>>>(d_matrix, d_result);

  GpuElapse(start, stop, elapsed, total);
  printf("column_reduce Time elapsed %f us\n", elapsed / ITERS * 1000);

  CHECK(
      hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipDeviceReset());

  // column reduction on host
  column_reduce_host(matrix, h_result);

  // // check result
  // check_result(result, h_result);

  // for(int i = 0; i < N; i++){
  //     printf("result[%d] = %f ", i, result[i]);
  // }

  free(matrix);
  free(result);
  free(h_result);
  hipFree(d_matrix);
  hipFree(d_result);

  return 0;
}