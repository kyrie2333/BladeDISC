#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// #define M 8192
// #define N 2560
// #define BLOCK_X 40
// #define BLOCK_Y 32
// #define TILE_SIZE 16

#define WARMUPS 100
#define ITERS 100

#define CHECK(call)                                          \
  {                                                          \
    const hipError_t error = call;                          \
    if (error != hipSuccess) {                              \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
      fprintf(stderr, "code: %d, reason: %s\n", error,       \
              hipGetErrorString(error));                    \
      exit(1);                                               \
    }                                                        \
  }

#define GpuElapse(start, stop, elapsed) \
  hipEventRecord(stop, 0);                    \
  hipEventSynchronize(stop);                  \
  hipEventElapsedTime(&elapsed, start, stop); 
  // total += elapsed;

// check result
// | (real - expected) / expected |
void check_result(float* host_ref, float* gpu_ref, int N) {
  double epsilon = 1.0E-4;
  bool match = 1;
  for (int i = 0; i < N; i++) {
  gpu_ref[i] /= 200;

    if (abs((host_ref[i] - gpu_ref[i]) / host_ref[i]) > epsilon) {
      match = 0;
      printf("Wrong!\n");
      printf("host %5.8f gpu %5.8f at index %d, error %5.8f\n", host_ref[i],
             gpu_ref[i], i, abs((host_ref[i] - gpu_ref[i]) / host_ref[i]));
      break;
    }
  }
  if (match) printf("Match\n");
}

// column reduction on host
void column_reduce_host(float* matrix, float* result, int M, int N) {
  for (int col = 0; col < N; col++) {
    float accum = 0.0;
    for (int row = 0; row < M; row++) {
      accum += matrix[row * N + col];
    }
    result[col] = accum;
  }
}

// colume reduction for a mtrix, 1D grid and 2D block
__global__ void column_reduce_trans(float* data_in, float* data_out, int M, int N, int tile_size) {
  __shared__ float sdata[32][8];

  int block_x = blockIdx.x % ((N + blockDim.x - 1) / blockDim.x);
  int block_y = blockIdx.x / ((N + blockDim.x - 1) / blockDim.x);

  int col_g = block_x * blockDim.x + threadIdx.x;

  // local reduction
    float accum = 0.0f;
  if (col_g < N) {
    for (int i = 0; i < tile_size; i++) {
      int row_g = i + threadIdx.y * tile_size + block_y * tile_size * blockDim.y;
      if (row_g < M && col_g < N)
        accum += data_in[row_g * N + col_g];
      else
        accum += 0.0f;
    }
  }else{
    sdata[threadIdx.x][threadIdx.y] = 0.0f;
  }
    sdata[threadIdx.x][threadIdx.y] = accum;
  __syncthreads();

  for (int stride = blockDim.y / 2; stride > 0; stride >>= 1) {
    if (threadIdx.y < stride) {
      sdata[threadIdx.x][threadIdx.y] +=
          sdata[threadIdx.x][threadIdx.y + stride];
    }
    __syncthreads();
  }

  // block level reduction
  if (col_g < N && threadIdx.y == 0) {
    atomicAdd(&data_out[col_g], sdata[threadIdx.x][0]);
  }
}

int main(int argc, char *argv[]) {

  if (argc != 6) {
    // printf("5 arguments required: M, N, BLOCK_X, BLOCK_Y, TILE_SIZE\n");
    exit(1);
  }

  int M = atoi(argv[1]);
  int N = atoi(argv[2]);
  int BLOCK_X = atoi(argv[3]);
  int BLOCK_Y = atoi(argv[4]);
  int TILE_SIZE = atoi(argv[5]);

  // allocate memory for matrix and result on host
  float* matrix = (float*)malloc(M * N * sizeof(float));
  float* result = (float*)malloc(N * sizeof(float));
  float* h_result = (float*)malloc(N * sizeof(float));

  // initialize matrix
  for (int i = 0; i < M * N; i++) {
    matrix[i] = (float)rand() / RAND_MAX;
    // matrix[i] = 1.0f;
  }

  // allocate memory for matrix and result on device
  float *d_matrix, *d_result;
  CHECK(hipMalloc((void**)&d_matrix, M * N * sizeof(float)));
  CHECK(hipMalloc((void**)&d_result, N * sizeof(float)));
  CHECK(hipMemcpy(d_matrix, matrix, M * N * sizeof(float),
                   hipMemcpyHostToDevice));
                   

  dim3 block(BLOCK_X, BLOCK_Y);
  int grid_y = (M + (TILE_SIZE * BLOCK_Y) - 1) / (TILE_SIZE * BLOCK_Y);
  dim3 grid((N + BLOCK_X - 1) / BLOCK_X * grid_y, 1);

  for (int i = 0; i < WARMUPS; ++i)
    column_reduce_trans<<<grid, block>>>(d_matrix, d_result, M, N, TILE_SIZE);

  // float total = 0.;
  float elapsed = 0.;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < ITERS; ++i)
    column_reduce_trans<<<grid, block>>>(d_matrix, d_result, M, N, TILE_SIZE);

  GpuElapse(start, stop, elapsed);
  printf("Time  %f us\n", elapsed / ITERS * 1000);

  CHECK(hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipDeviceReset());

  // column reduction on host
  column_reduce_host(matrix, h_result, M, N);

  // check result
  check_result(h_result, result, N);

  free(matrix);
  free(result);
  free(h_result);
  hipFree(d_matrix);
  hipFree(d_result);

  return 0;
}
