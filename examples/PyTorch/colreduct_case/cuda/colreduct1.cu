#include <hip/hip_runtime.h>
// #include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// #define M 8192
// #define N 2560
// #define BLOCK_X 256
// #define TILE_SIZE 128

#define WARMUPS 100
#define ITERS 100

#define CHECK(call)                                          \
  {                                                          \
    const hipError_t error = call;                          \
    if (error != hipSuccess) {                              \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
      fprintf(stderr, "code: %d, reason: %s\n", error,       \
              hipGetErrorString(error));                    \
      exit(1);                                               \
    }                                                        \
  }

#define CpuElapse(base, start) \
  base += ((double)(clock() - start)) * 1000 / CLOCKS_PER_SEC;

#define GpuElapse(start, stop, elapsed, total) \
  hipEventRecord(stop, 0);                    \
  hipEventSynchronize(stop);                  \
  hipEventElapsedTime(&elapsed, start, stop); \
  total += elapsed;

// check result
// | (real - expected) / expected |
void check_result(float* host_ref, float* gpu_ref, int N) {
  double epsilon = 1.0E-4;
  bool match = 1;
  for (int i = 0; i < N; i++) {
  gpu_ref[i] /= 200;

    if (abs((host_ref[i] - gpu_ref[i]) / host_ref[i]) > epsilon) {
      match = 0;
      printf("wrong!\n");
      printf("host %5.8f gpu %5.8f at index %d, error %5.8f\n", host_ref[i],
             gpu_ref[i], i, abs((host_ref[i] - gpu_ref[i]) / host_ref[i]));
      break;
    }
  }
  if (match) printf("Match\n");
}
// column reduction on host
void column_reduce_host(float* matrix, float* result, int M, int N) {
  for (int col = 0; col < N; col++) {
    float accum = 0.0;
    for (int row = 0; row < M; row++) {
      accum += matrix[row * N + col];
    }
    result[col] = accum;
  }
}

// colume reduction for a mtrix, 1D grid and 1D block
// each thread will reduce a column in a block
// each accum of a block is reduce by atomicAdd
__global__ void column_reduce(float* data_in, float* data_out, int M, int N, int tile_size) {
  // get the column index
  // int col = blockIdx.x * blockDim.x + threadIdx.x;
  int block_x = blockIdx.x % ((N + blockDim.x - 1) / blockDim.x);
  int block_y = blockIdx.x / ((N + blockDim.x - 1) / blockDim.x);
  int col = block_x * blockDim.x + threadIdx.x;

  float accum = 0.0;

  for (int i = 0; i < tile_size; i++) {
    int row = block_y * tile_size + i;
    if (row < M) accum += data_in[row * N + col];
  }

  atomicAdd(&data_out[col], accum);
}

int main(int argc, char* argv[]) {

  if(argc != 5){
    // printf("4 arguments required: M N BLOCK_X TILE_SIZE\n");
    exit(1);
  }
  int M = atoi(argv[1]);
  int N = atoi(argv[2]);
  int BLOCK_X = atoi(argv[3]);
  int TILE_SIZE = atoi(argv[4]);

  // allocate memory for matrix and result on host
  float* matrix = (float*)malloc(M * N * sizeof(float));
  float* result = (float*)malloc(N * sizeof(float));    // gpu result
  float* h_result = (float*)malloc(N * sizeof(float));  // validation

  // initialize matrix
  for (int i = 0; i < M * N; i++) {
    matrix[i] = (float)rand() / RAND_MAX;
    // matrix[i] = 1.0;
  }

  // allocate memory for matrix and result on device
  float *d_matrix, *d_result;
  CHECK(hipMalloc((void**)&d_matrix, M * N * sizeof(float)));
  CHECK(hipMalloc((void**)&d_result, N * sizeof(float)));

  CHECK(hipMemcpy(d_matrix, matrix, M * N * sizeof(float),
                   hipMemcpyHostToDevice));

  dim3 block(BLOCK_X, 1);
  int grid_y = (M + TILE_SIZE - 1) / TILE_SIZE;
  dim3 grid((N + BLOCK_X - 1) / BLOCK_X * grid_y, 1);

  for (int i = 0; i < WARMUPS; ++i)
    column_reduce<<<grid, block>>>(d_matrix, d_result, M, N, TILE_SIZE);

  float total = 0.;
  float elapsed = 0.;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < ITERS; ++i)
    column_reduce<<<grid, block>>>(d_matrix, d_result, M, N, TILE_SIZE);

  GpuElapse(start, stop, elapsed, total);
  printf("Time %f us\n", elapsed / ITERS * 1000);

  CHECK(hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipDeviceReset());

  // column reduction on host
  column_reduce_host(matrix, h_result, M, N);

  // check result
  check_result(h_result, result, N);

  free(matrix);
  free(result);
  free(h_result);
  hipFree(d_matrix);
  hipFree(d_result);

  return 0;
}